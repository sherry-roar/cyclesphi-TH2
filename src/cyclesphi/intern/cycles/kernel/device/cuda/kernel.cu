
#include <hip/hip_runtime.h>
/* SPDX-License-Identifier: Apache-2.0
 * Copyright 2011-2022 Blender Foundation */

/* CUDA kernel entry points */

#ifdef __CUDA_ARCH__

#  include "kernel/device/cuda/compat.h"
#  include "kernel/device/cuda/config.h"
#  include "kernel/device/cuda/globals.h"

#  if defined(WITH_CUDA_CPUIMAGE)
#    include "kernel/device/cpu/image.h"
#  else
#    include "kernel/device/gpu/image.h"
#  endif

#  include "kernel/device/gpu/kernel.h"

#endif
